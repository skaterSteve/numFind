#include "hip/hip_runtime.h"

/*			  KD Tree Implementation
 *			       by Steven Wu
 *				3-29-2011
 * 	  
 *	An ideal tree is calculated with a given # of levels.
 *	The tree is built iteratively, and all N points are used to fill the tree.	
 *	After the tree is built, a new point p0 is placed in the grid.
 *	A recursive function is used to find the nearest point.
 *
 */
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <cstdlib>
#include <iostream>
//#include <stdio.h>
//#include <stdlib.h>
#include <math.h>

static int dimsize = 2000;

struct Point {
    double x;
    double y;
    struct Point *left, *right;
};

/********************************************************
*	Return int value 2^level			*
*	Used to calculate  number of N for ideal tree	*
*	and # times to run iterations at each level	*
*********************************************************/
int twotothe( int level ) { 		
// a function to return int value 2^level
    int N =1;
    while ( level > 0 ) {
	N *=2;
	level--;
    }
    return (N);
}


/********************************************************
*    Print specified portions of array of struct Point	*
*********************************************************/
void printArray( int i_start, int i_end, struct Point arr[] ) {
    int i;
    for( i = i_start; i < i_end; i++ ) {
	printf(" %5d   %6.3f    %6.3f\n", i, arr[i].x, arr[i].y );
    }
}


/********************************************************************************
* 		Place N random Point(s) within dimsize box 			*
*********************************************************************************/

struct Point randoms( int N, struct Point *arr ) { 
    int i; 				    
 //   srand(time(NULL));
    int seed;
    seed = 11.7777;
    srand(seed);

    for( i = 0; i < N; i++) {
	arr[i].x = (double)dimsize * ( rand() % RAND_MAX  ) / RAND_MAX;
	arr[i].y = (double)dimsize * ( rand() % RAND_MAX  ) / RAND_MAX;
    }
    return *arr;
}

/********************************************************************************
*  		"Bubble Sort" is an O( N^2 ) Sorting Routine.			*
*  		Sort by x- or y- direction as determined by "dir" 		*
*   										*
*   It is seen to be very slow at large N ( depth 18 is N = 524,287 ) 		*
*   Computation time could be greatly reduced by implementing CUDA 		*
*   for large N.  For smaller N, sorting on CPU may be quicker			*
*   by not transferring data back and forth between CPU and GPU.		*
*										*
*   Empirically, sorting for depth = 14 (N ~ 32000) is acceptable (~7sec).	*
*   It would be most efficient to use a CUDA routine for depth > 14.		*
*   for N = 65535 (depth=15), computation time is (~1 min 10 sec)		*
*********************************************************************************/
struct Point bubbleSort( int i_start, int i_end, struct Point *array, int dir ) {	
    double tempx, tempy;
    int i, j;

    if( dir % 2 == 0 ) { 					//X-sort
	for ( i = i_start; i < i_end; i++ ) {
   	    for ( j = i_start; j < i_end-i-1+i_start; j++ ) {
		if( array[j].x > array[j+1].x )  {
		    tempx = array[j].x;
		    tempy = array[j].y;
		    array[j].x = array[j+1].x;
		    array[j].y = array[j+1].y;
		    array[j+1].x = tempx;
		    array[j+1].y = tempy;
	    	}
	    }
    	}
    }
    else {
    	for ( i = i_start; i < i_end; i++ ) {			//Y-sort
   	    for ( j = i_start; j < i_end-i-1+i_start; j++ ) {
	    	if( array[j].y > array[j+1].y )  {
		    tempx = array[j].x;
		    tempy = array[j].y;
		    array[j].x = array[j+1].x;
		    array[j].y = array[j+1].y;
		    array[j+1].x = tempx;
		    array[j+1].y = tempy;
	    	}
	    }
    	}
    }
    return *array;
}


float fillArrays( int i_start, int i_end, struct Point *a, float *tempx, float *tempy ) {
    // transfer struct members into temp arrays & reset index
    for( int i = i_start; i < i_end; i++ ) {
	tempx[i] = (a+i)->x;
	tempy[i] = (a+i)->y;

//	std::cout << i << "  " << tempx[i] << "  " << tempy[i] << std::endl;
    }
    return *tempx, *tempy;
}

int resetIndex( int i_start, int i_end, int *index ) {
    for( int i = i_start; i < i_end; i++ ) {
	index[i] = i;
    }
    return *index;
}


struct Point updateStruct( int i_start, int i_end, struct Point *a, float *tempx, 
			   float *tempy, int *index, int level ) {
    //update struct with sorted values in x or y dir
    if( level % 2 == 0 ) {
	for(int i = i_start; i < i_end; i++)  {
	    (a+i)->x = tempx[ i ];
	    (a+i)->y = tempy[ index[i] ];
	}
    }
    else if( level % 2 == 1 ) {		
	for(int i = i_start; i < i_end; i++)  {
	    (a+i)->x = tempx[ index[i] ];
	    (a+i)->y = tempy[ i ];
	}
    }
    return *a;
}

/************************************************************************
*	BINARY TREE OPERATIONS						*
*	http://cslibrary.stanford.edu/110/BinaryTrees.html 		*
*									*
* 	Original code author: Nick Parlante.				*
*  	Modified by Steven Wu.						*
*************************************************************************/			

/*-----------------------------------------------------------------------
 Helper function that allocates a new node 
 with the given data and NULL left and right Pointers.
------------------------------------------------------------------------*/
struct Point* NewNode( struct Point *p ) {						
    struct Point* parent;
    parent = (struct Point*)malloc(sizeof(struct Point));

    parent = p;
    parent->left = NULL;
    parent->right = NULL;

    return(parent);
}

/*-----------------------------------------------------------------------
 Given a binary search tree, and x & y coords, the new node is placed 
 in the tree correctly by alternating x- or y- dir according to depth.
 Starts at root with x condition, is (iterated) using "tick."		
-------------------------------------------------------------------------*/

struct Point *insert( int tick, struct Point* parent, struct Point* child ) { 				

    if (parent == NULL )		//  1.  If the tree is empty,
    	return  NewNode(child) ;	//      return a new, single node
 
    else if( tick % 2 == 1 )  {		//  2.  Otherwise, recur down the tree

     	if ( child->x <= parent->x )	
	    parent->left = insert( tick + 1, parent->left, child );
    	else 	
	    parent->right = insert( tick + 1, parent->right, child );

    	return(parent); // return the (unchanged) node pointer
    }
    else if( tick % 2 == 0 )  {

     	if ( child->y <= parent->y )	
	    parent->left = insert( tick + 1, parent->left, child );
    	else 
	    parent->right = insert( tick + 1, parent->right, child );

    	return(parent); // return the (unchanged) node pointer
    }
    return;
}

	
/*****************************************************************************
*	A recursive function to find the leaf closest to the new point p.    * 
******************************************************************************/

struct Point simple_fn( int tick, struct Point p, struct Point *parent ) {

    if( parent->left == NULL && parent->right == NULL )    //return if leaf	
	return *parent;					

    else if( tick % 2 == 0)  {			// x- dim comparison
	tick++;
	if( p.x <= parent->x )
	    parent = parent->left;
    	else 
	    parent = parent->right;
	return simple_fn( tick, p, parent );	//call function, with tick+1
    }

    else if( tick % 2 == 1 )  {			// y- dim comparison
	tick++;
	if( p.y <= parent->y )
	    parent = parent->left;
    	else 
	    parent = parent->right;
	return simple_fn( tick, p, parent );
    }
    return;
}



  //======================================================================//
 //	main() starts here					 	 //
//======================================================================//

main()  
{
    int i;				// array index
    int N = 1; 				// Number of N for the level = 0 case
    int level_tot = 26;			// Enter the total number of levels
    int level = level_tot;              // Initialize level counter
    int i_start, i_end;			// Index start/end of sub-arrays
    int median;				// Median value chosen multiple times 
    int tick;				// Used to ensure leaf runs through 
					// conditions starting at root
    double distx, disty, distance;

   while ( level > 0 )  {		// For a given level,
	N += twotothe( level );         // calculate N needed for
	level--;			// balanced tree
    }

    int N_tot = N;			// Retain N value for after tree is built

    struct Point *a;  			// one Point of x and y coords
    a = (struct Point*)malloc( N * sizeof( *a ) );     // Allocate memory for N Points

    struct Point* root;
    root = NULL;

    float *tempx, *tempy;
    tempx = (float*)malloc( N * sizeof( *tempx) );
    tempy = (float*)malloc( N * sizeof( *tempy) );

    int *index;
    index = (int*)malloc( N * sizeof( *index ) ); 


    randoms( N, a );    		// place N random pts in box

//  printArray(0, N_tot, a);

    level = 0;				// Start at root


/********************************************************************************
*   Sort initial array in x- dir, find median, and place coords in root.	*
*   Split into two arrays at the median, 					*
*   sort each array in y- dir and place each median in tree.			*
*   Switching off between x- and y- conditions,					*
*   repeat until while() condition is met, and place coords correctly in tree.  *
*********************************************************************************/
	
    while( N >=3 ) {

//	Uncomment print statements to check points at every level
	printf("\nLEVEL %d\n", level);

   	for( i = 0; i < twotothe(level); i++ )  {         

	    i_start = i * ( N + 1 );
	    i_end   = i_start + N;    
	    printf("i = %d to  i < %d:\n", i_start, i_end);

	    fillArrays( i_start, i_end, a, tempx, tempy );
	    resetIndex( i_start, i_end, index );

//	    bubbleSort( i_start, i_end, a, level );
 
	    if( level % 2 == 0 )	
		thrust::stable_sort_by_key( tempx + i_start, tempx + i_end, index + i_start); 
	    else 	
		thrust::stable_sort_by_key( tempy + i_start, tempy + i_end, index + i_start); 

    	    updateStruct( i_start, i_end, a, tempx, tempy, index, level );
	    printArray( i_start, i_end, a );			

	    median = N/2+i*(N+1);
	    printf("median = %d\n", median );

	    tick = 1;		// reset the ticker before each call to insert()
	    root = insert( tick, root, ( a + median ) );
	}

    	level++;
    	N/=2;
    }


/************************************************************************
*		Find the closest point to p0.				*
*	If the distance to a leaf's parent is closer than the leaf,	*
*	this function call does not return the closest point		*
*************************************************************************/


    struct Point p0;

    p0.x = 3;		// Arbitrarily chosen points within dimsize.
    p0.y = 18;		// Round numbers avoid running into same value in grid.

    for( i = 0; i < N_tot; i++ ) {
	distx = p0.x - (a+i)->x;
	distx *= distx;
	disty = p0.y - (a+i)->y;
	disty *= disty;
	distance = sqrt(distx + disty);
	printf("%d  ( %f   %f )     %f\n", i, (a+i)->x, (a+i)->y, distance);
    }
    printf("**************************************\n");    
	
    level = 0;

    *root = simple_fn( level, p0, root ) ;
    printf("Closest point to p0: %f %f\n", root->x, root->y );

    printf("N = %d\n", N_tot);       	 
    printf("total levels in tree: %d\n", level_tot);

    free( a );
    free( tempx );
    free( tempy );
    free( index );

    return 0;

}



